#include <stdint.h>
#include <unistd.h>

#include <png.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define rel params[0]
#define img params[1]
#define scl params[2]

__device__ void writeHSV(uint8_t *pixel, int theta) {
    unsigned char region, remainder, q, t;
    region = theta / 43;
    remainder = (theta - (region * 43)) * 6;
   
    q = (255 * (255 - ((255 * remainder) >> 8))) >> 8;
    t = (255 * (255 - ((255 * (255 - remainder)) >> 8))) >> 8;

    switch (region) {
        case 0:
            *pixel++ = 255;
            *pixel++ = t;
            *pixel++ = 0;
            return;
        case 1:
            *pixel++ = q;
            *pixel++ = 255;
            *pixel++ = 0;
            return;
        case 2:
            *pixel++ = 0;
            *pixel++ = 255;
            *pixel++ = t;
            return;
        case 3:
            *pixel++ = 0;
            *pixel++ = q;
            *pixel++ = 255;
            return;
        case 4:
            *pixel++ = t;
            *pixel++ = 0;
            *pixel++ = 255;
            return;
        default:
            *pixel++ = 255;
            *pixel++ = 0;
            *pixel++ = q;
            return;
    }
}

__global__ void euclid (uint8_t *gpu, double *params, int streamNumber ) {
    int index, pos;
    int c, t;
    uint32_t x, y; 

    index = streamNumber * 65536 + threadIdx.x * 256;
    for (pos = 0; pos < 256; pos++) {
        x = (uint32_t) (((rel + 2.0) + (double) (.5 + (index % 1024)) * scl) * 1048576);
        y = (uint32_t) (((img + 2.0) + (double) (.5 + (index / 1024)) * scl) * 1048576);
        c = 0;
        t = 1;
        while (1) {
            if (x > y) {
                x -= y;
                c++;
            } else if (y > x) {
                y -= x;
            } else {
                break;
            }
            t++;
            if (t > 1000) break;
        }

        uint8_t *pixel = (gpu + index++ * 3);
        *pixel++ = (255 * c) / t;
        *pixel++ = (255 * c) / t;
        *pixel++ = (255 * c) / t;
    }
}

__global__ void mandelbrot (uint8_t *gpu, double *params, int streamNumber ) {
    int index, c, pos;
    double cr, ci, zr, zi, t;

    index = streamNumber * 65536 + threadIdx.x * 256;
    for (pos = 0; pos < 256; pos++) {
        c = 0;
        cr = rel + (double) (.5 + (index % 1024)) * scl / 1024.0;
        ci = img + (double) (.5 + (index / 1024)) * scl / 1024.0;
        zr = cr;
        zi = ci;

        while (++c < 1000 && zr * zr + zi * zi < 4) {
            t = zr;
            zr = zr * zr - zi * zi + cr;
            zi = 2 * t * zi + ci;
        }
    
        uint8_t *pixel = (gpu + index * 3);
        if (c == 1000) {
            *pixel++ = 0;
            *pixel++ = 0;
            *pixel++ = 0;
        } else {
            writeHSV(pixel, c);
        }
        index ++;
    }
}

// GPU variables
double *gpu_params;
uint8_t *gpu; 

// Host variables
hipStream_t streams[16];
double params[3];
png_byte ** row_pointers;
void (*kernel) (uint8_t *, double *, int);


// reads parameters from stdin and writes them to params array
// initializes rel, img, and scl macros
void readParams() {
    rel = -2.0;
    img = -2.0;
    scl = 4.0;
    char c = getchar();
    switch (c) {
        case 'm':
            kernel = mandelbrot;
            break;
        default:
            kernel = euclid;
    }
    while ((c = getchar()) != '@') {
        scl /= 3.0;
        switch (c) {
            case '3':
            case '6':
            case '9':
                rel += scl;
            case '2':
            case '5':
            case '8':
                rel += scl;
            default:
                break;
        }
        switch (c) {
            case '7':
            case '8':
            case '9':
                img += scl;
            case '4':
            case '5':
            case '6':
                img += scl;
            default:
                break;
        }
    }
}

// begins computation
void computeKernel() {
    // setup params
    hipMemcpy( gpu_params, params, 3 * sizeof(double), hipMemcpyHostToDevice);

    // initialize streams
    int i, r;
    for (i = 0; i < 16; i++) {
        hipStreamCreate((streams + i));
    }

    // execute kernels in the streams
    for (i = 0; i < 16; i++) {
        kernel<<<1, 256, 0, streams[i]>>>( gpu, gpu_params, i );
    }

    // setup asynchronous memory copy after completion
    for (i = 0; i < 16; i++) {
        for (r = 0; r < 64; r++) {
            hipMemcpyAsync(row_pointers[64 * i + r], (gpu + i * 65536 * 3 + r * 1024 * 3), sizeof(uint8_t) * 1024 * 3, hipMemcpyDeviceToHost, streams[i]);
        }
    }

    hipDeviceSynchronize();
}

extern void writePngOutput();

int main(int argc, char **argv) {

    // Initialize memory
    hipMalloc( (void**)  &gpu, 1024 * 1024 * sizeof(uint8_t) * 3 );
    hipMalloc( (void**)  &gpu_params, 3 * sizeof(double) );

    row_pointers = (png_byte **) malloc (1024 * sizeof (png_byte *));
    for (int y = 0; y < 1024; y++) {
        row_pointers[y] = (png_byte *) malloc (sizeof (uint8_t) * 1024 * 3);
    }

   
    // do the process
    while (1) {
        readParams();
        computeKernel();
        writePngOutput();
    }

}

size_t pngBufferFill = 0;
extern void writeFn(png_structp png_ptr, png_bytep data, uint32_t size);
extern void flushFn(png_structp png_ptr);

void writePngOutput() {

    png_structp png_ptr = png_create_write_struct (PNG_LIBPNG_VER_STRING, NULL, NULL, NULL);
    png_infop   info_ptr = png_create_info_struct (png_ptr);
    
    png_set_IHDR (png_ptr,
            info_ptr,
            1024,                           // width
            1024,                           // height
            8,                              // depth
            PNG_COLOR_TYPE_RGB,
            PNG_INTERLACE_NONE,
            PNG_COMPRESSION_TYPE_DEFAULT,
            PNG_FILTER_TYPE_DEFAULT);

    png_set_write_fn(png_ptr, NULL, (png_rw_ptr) writeFn, (png_flush_ptr) flushFn);

    png_init_io (png_ptr, stdout);
    png_set_rows (png_ptr, info_ptr, row_pointers);
    png_write_png (png_ptr, info_ptr, PNG_TRANSFORM_IDENTITY, NULL);

    write(2, &pngBufferFill, 4);
    pngBufferFill = 0;

    png_destroy_write_struct (&png_ptr, &info_ptr);
}

void writeFn(png_structp png_ptr, png_bytep data, uint32_t size) {
    write(1, data, size);
    pngBufferFill += size;
}

void flushFn(png_structp png_ptr) {
    fflush(stdout);
}
